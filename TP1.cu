#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <cstdlib> //pour le rand

/* on répertorie comme dans un fichier h*/
void MatrixInit(float *M, int n, int p);
void MatrixPrint(float *M, int n, int p);
__host__ void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p);
__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p);
void MatrixMult(float *M1, float *M2, float *Mout, int n);
__global__ void cudaMatrixMult(float *M1, float *M2, float *Mout, int n);

void MatrixInit(float *M, int n, int p){
    for (int i = 0; i < n*p; i++){
        M[i] = (rand() % 3) -1;
    }
}

void MatrixPrint(float *M, int n, int p){
    for (int i = 0; i < n*p ; i++){
        if((i+1)%n ==0){
            printf("%f\n",M[i]);
        }else{
            printf("%f ",M[i]);
        }
    }
}

__host__ void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
    for (int i = 0; i < n*p; i++){
        Mout[i] = M1[i] + M2[i];
    }
}

__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
    int col = blockIdx.x; //premier élément dans l'appel à la fonction
    int row = threadIdx.x; //deuxième élément dans l'appel à la fonction
    Mout[row*n + col] = M1[row*n + col] + M2[row*n + col];
    
    printf("Hello\n");
    //pour lancer le thread, on fait cudaMatrixAdd<<<n,p>>>(M1,M2,Mout,n,p)
}

void MatrixMult(float *M1, float *M2, float *Mout, int n){
    for(int row = 0; row < n; row++){
        for(int col = 0; col < n; col++){
            int tmp = 0;
            for( int i = 0; i < n; i++){
                tmp += M1[row * n + i] * M2[i * n + col];
            }
            //Write back the result
            Mout[row * n + col] = tmp;
        }
    }
}

__global__ void cudaMatrixMult(float *M1, float *M2, float *Mout, int n){
    int col = blockIdx.x; //premier élément dans l'appel à la fonction
    int row = threadIdx.x; //deuxième élément dans l'appel à la fonction
    if(row < n && col < n){
        //Accumulate a partial result
        int tmp = 0;
        for( int i = 0; i < n; i++){
            tmp += M1[row * n + i] * M2[i * n + col];
        }
        //Write back the result
        Mout[row * n + col] = tmp;
    }
}


int main(){
    
    //tailles des matrices
    int n = 2;
    int p = 3;
    
    //taille en mémoire des matrices pour l'addition
    const int ARRAY_SIZE = n*p;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
    
    //taille en mémoire des matrices pour la multiplication
    const int ARRAY_SIZE2 = n*n;
    const int ARRAY_BYTES2 = ARRAY_SIZE2 * sizeof(float);
    
    // variables host + allocation de mémoire sur CPU
    // pour l'addition
    float *a, *b, *out;
    a   = (float*)malloc(ARRAY_BYTES);
    b   = (float*)malloc(ARRAY_BYTES);
    out = (float*)malloc(ARRAY_BYTES);
    //pour la multiplication
    float *a2, *b2, *out2;
    a2   = (float*)malloc(ARRAY_BYTES2);
    b2   = (float*)malloc(ARRAY_BYTES2);
    out2 = (float*)malloc(ARRAY_BYTES2);
    
    //initialisation des matrices tests:
    //pour l'addition
    MatrixInit(a, n, p);
    MatrixInit(b, n, p);
    //pour la multiplication
    MatrixInit(a2, n, n);
    MatrixInit(b2, n, n);
    
    //calcul sur cpu :
    //addition
    MatrixAdd(a, b, out, n, p);
    //multiplication
    MatrixMult(a2, b2, out2, n);
    
    //affichage :
    printf("Matrice a :\n");
    MatrixPrint(a, n, p);
    printf("Matrice b :\n");
    MatrixPrint(b, n, p);
    printf("Matrice a+b sur cpu :\n");
    MatrixPrint(out, n, p);
    printf("Matrice a2 :\n");
    MatrixPrint(a2, n, n);
    printf("Matrice b2 :\n");
    MatrixPrint(b2, n, n);
    printf("Matrice a2*b2 sur cpu :\n");
    MatrixPrint(out2, n, n);
    
    //calcul GPU
    // variables device + allocation de mémoire sur GPU
    float *d_a, *d_b, *d_out, *out1, *d_a2, *d_b2, *d_out2, *out22;
    hipMalloc((void **) &d_a, ARRAY_BYTES);
    hipMalloc((void **) &d_b, ARRAY_BYTES);
    hipMalloc((void **) &d_out, ARRAY_BYTES);
    hipMalloc((void **) &d_a2, ARRAY_BYTES2);
    hipMalloc((void **) &d_b2, ARRAY_BYTES2);
    hipMalloc((void **) &d_out2, ARRAY_BYTES2);
    out1 = (float*)malloc(ARRAY_BYTES);
    out22 = (float*)malloc(ARRAY_BYTES2);
    
    //transfert de données CPU vers GPU pour le calcul sur gpu
    hipMemcpy(d_a, a, ARRAY_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, ARRAY_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_a2, a2, ARRAY_BYTES2, hipMemcpyHostToDevice);
    hipMemcpy(d_b2, b2, ARRAY_BYTES2, hipMemcpyHostToDevice);
    
    //calcul sur gpu
    //addition : on fait n blocks de p threads
    printf("Ici\n");
    cudaMatrixAdd<<<n,p>>>(d_a,d_b,d_out,n,p);
    //multiplication : on fait n blocks de n threads
    cudaMatrixMult<<<n,n>>>(d_a2,d_b2,d_out2,n);
    
    //récupération des données du GPU vers le CPU
    hipMemcpy(out1, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(out22, d_out2, ARRAY_BYTES2, hipMemcpyDeviceToHost);
    
    //affichage du résultat
    printf("Matrice a+b sur gpu :\n");
    MatrixPrint(out1, n, p);
    printf("Matrice a2*b2 sur gpu :\n");
    MatrixPrint(out22, n, n);
    
    //libération des ressources 
    // sur GPU :
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_a2);
    hipFree(d_b2);
    hipFree(d_out2);
    // sur CPU
    free(a);
    free(b);
    free(a2);
    free(b2);
    free(out);
    free(out1);
    free(out2);
    free(out22);
    
    // This call waits for all of the submitted GPU work to complete
    hipDeviceSynchronize();
    
    return 0;
}
