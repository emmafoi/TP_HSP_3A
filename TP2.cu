#include "hip/hip_runtime.h"
//Partie 2

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdlib>

void MatrixInitRand(float *M, int n);
void MatrixInitZero(float *M, int n);
void MatrixPrint(float *M, int n);
__global__ void cudaConv(float *E, float *F, float *S);
__global__ void cudaMoyen2(float *E, float *F, int n);

// 3.1

void MatrixInitRand(float *M, int n){
    for (int i = 0; i < n; i++){
        M[i] = (float)(rand()%1000)/1000 ; 
        //flottant entre 0 et 1 de précision 10⁻3
    }
}

void MatrixInitZero(float *M, int n){
    for (int i = 0; i < n; i++){
        M[i] = 0 ; 
        //flottant entre 0 et 1 de précision 10⁻3
    }
}

void MatrixPrint(float *M, int n){
    for (int i = 0; i < n ; i++){
        if((i+1)%n ==0){
            printf("%f\n",M[i]);
        }else{
            printf("%f ",M[i]);
        }
    }
}

// 3.2

/* A REPRENDRE*/
__global__ void cudaConv(float *E, float *F, float *S){
    int idx = threadIdx.x;
    S[idx] = E[idx] * F[idx];
}

__global__ void cudaMoyen2(float *E, float *S, int n){
    // n = taille d'une ligne de E (et aussi d'une colonne)
    
    //1er élément du 1er dim3 = nombre matrices 2D de E
    int nb_mat = blockIdx.x;
    //nb_mat * taille d'une matrice de S (= taille du shift dans l'indice de S):
    int shift_S = nb_mat * n/2 * n/2 ;
    //nb_mat * taille d'une matrice de E (= taille du shift dans l'indice de E):
    int shift_E = nb_mat * n * n ;
    
    //2e élément du 1er dim3 = nombre de colonnes/2 de E = nombre de col de S
    int output_col = blockIdx.y; 
    //2e dim3 (contient 1 seul élément) = nombre de lignes/2 de E = nombre de lignes de S
    int output_row = threadIdx.x;
    
    //on se déplace de 2 en 2 dans les matrices d'entrée
    int input_col = 2 * output_col;
    int input_row = 2 * output_row;
    
    //Calcul de S en fonction de E :
    S[shift_S + output_row * n + output_col] = (float)(( E[shift_E + input_row * n + input_col] + E[shift_E + (input_row+1) * n + input_col] + E[shift_E + input_row * n + (input_col+1)] + E[shift_E + (input_row+1) * n + (input_col+1)] )/4);
}


int main(){
    
    // 3.1 
    
    //matrice raw_data
    int n1 = 32;
    const int ARRAY_SIZE1 = n1*n1;
    const int ARRAY_BYTES1 = ARRAY_SIZE1 * sizeof(float);
    
    //matrice C1_data
    int n21 = 28;
    int n22 = 6;
    const int ARRAY_SIZE2 = n21*n21*n22;
    const int ARRAY_BYTES2 = ARRAY_SIZE2 * sizeof(float);
    
    //matrice S1_data : issue du sous-échantillonnage de facteur 2 de C1_data
    int n31 = 14;
    int n32 = 6;
    const int ARRAY_SIZE3 = n31*n31*n32;
    const int ARRAY_BYTES3 = ARRAY_SIZE3* sizeof(float);
    
    //matrice C1_kernel : 6 noyaux de conv de taille 5x5
    int n41 = 5;
    int n42 = 6;
    const int ARRAY_SIZE4 = n41*n41*n42;
    const int ARRAY_BYTES4 = ARRAY_SIZE4 * sizeof(float);
    
    //allocation de mémoire pour les matrices sur CPU
    float *raw_data, *C1_data, *S1_data, *C1_kernel;
    raw_data = (float*)malloc(ARRAY_BYTES1);
    C1_data = (float*)malloc(ARRAY_BYTES2);
    S1_data = (float*)malloc(ARRAY_BYTES3);
    C1_kernel = (float*)malloc(ARRAY_BYTES4);
    
     
    //initialisation :
    MatrixInitRand(raw_data, ARRAY_SIZE1);
    MatrixInitZero(C1_data, ARRAY_SIZE2);
    MatrixInitZero(S1_data, ARRAY_SIZE3);
    MatrixInitRand(C1_kernel, ARRAY_SIZE4);
    
    // pour tester :
    //MatrixPrint(C1_data, n21* n21* n22);
    
    
    // 3.2
       
    //allocation de mémoire sur GPU
    float *d_raw_data, *d_C1_data, *d_S1_data, *d_C1_kernel;
    hipMalloc((void **) &d_raw_data, ARRAY_BYTES1);
    hipMalloc((void **) &d_C1_data, ARRAY_BYTES2);
    hipMalloc((void **) &d_S1_data, ARRAY_BYTES3);
    hipMalloc((void **) &d_C1_kernel, ARRAY_BYTES4);
    
    //transfert de données pour le calcul sur gpu
    //entrée:
    hipMemcpy(d_raw_data, raw_data, ARRAY_BYTES1, hipMemcpyHostToDevice);
    //sortie:
    hipMemcpy(d_C1_data, C1_data, ARRAY_BYTES2, hipMemcpyHostToDevice);
    //filtre:
    hipMemcpy(d_C1_kernel, C1_kernel, ARRAY_BYTES4, hipMemcpyHostToDevice);
    
    // Layer 2 : convolution
    cudaConv<<<n21, n21, n22>>>(d_raw_data,d_C1_data, d_C1_kernel);
    
    //récupération des données sur le cpu
    hipMemcpy(C1_data, d_C1_data, ARRAY_BYTES2, hipMemcpyDeviceToHost);
    
    
   // Layer 3 : moyenneur
    dim3 my_blocks (n32, n31, 1) // taille = 6 * 28, on préfère regrouper comme ça
    //plutôt que 28*28 qui sera + gros 
    cudaMoyen2<<< my_blocks, n31>>>(d_C1_data,d_S1_data, n31);
    //ici, n32 = blockId.x et n31 = blockId.y pour se repérer dans la fonction
    
    
    
    //libération des ressources 
    hipFree(d_raw_data);
    hipFree(d_C1_data);
    hipFree(d_S1_data);
    hipFree(d_C1_kernel);
    
    free(raw_data);
    free(C1_data);
    free(S1_data);
    free(C1_kernel);
    
    // This call waits for all of the submitted GPU work to complete
    hipDeviceSynchronize();
    
    return 0;
}
